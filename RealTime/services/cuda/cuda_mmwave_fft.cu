#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <math.h>
#include <fstream>
#include <arpa/inet.h>
#include <unistd.h>
#include <thread>
#include <vector>

#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime_api.h>
#include <helper_string.h>
#include <unistd.h>
#include <sys/types.h>
#include <sys/wait.h>

#define CHIRP_LEN 256
#define NUM_CHIRPS 128
#define NUM_CHANNELS 4
#define RANGE_SIZE CHIRP_LEN
#define DOPPLER_SIZE NUM_CHIRPS
#define CLIENT_PORT 4095
#define FRAME_SIZE (NUM_CHANNELS * CHIRP_LEN * NUM_CHIRPS)
#define IN_SERVER_PORT 4097
#define IN_SERVER_IP_ADDRESS "192.168.33.28"
#define BUFFER_SIZE 65536
#define NO_OF_FRAMES = 1

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
int client_sockfd = 0;
struct sockaddr_in out_server_addr;
struct sockaddr_in in_server_addr;



/*
__device__ int g_uids = 0;

__global__ void cdp_kernel(int max_depth, int depth, int thread, int parent_uid)
{
    // We create a unique ID per block. Thread 0 does that and shares the value with the other threads.
    __shared__ int s_uid;

    if (threadIdx.x == 0)
    {
        s_uid = atomicAdd(&g_uids, 1);
    }

    __syncthreads();


    // We print the ID of the block and information about its parent.
    //print_info(depth, thread, s_uid, parent_uid);

    // We launch new blocks if we haven't reached the max_depth yet.
    if (++depth >= max_depth)
    {
        return;
    }

    cdp_kernel<<<gridDim.x, blockDim.x>>>(max_depth, depth, threadIdx.x, s_uid);
}

*/


__global__ void averageChannels(short *input, float2 *output) {
	
	printf("\nEntered Average channels"); 
	
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
	printf("\nCalculating average : %d %d %d %d", idx, blockIdx.x, blockDim.x, threadIdx.x );

    if (idx < CHIRP_LEN * NUM_CHIRPS) {

		        
		float2 sum = {0, 0};
        for (int ch = 0; ch < NUM_CHANNELS; ch++) {
            //sum.x += input[idx + ch * CHIRP_LEN * NUM_CHIRPS].x;
            //sum.y += input[idx + ch * CHIRP_LEN * NUM_CHIRPS].y;
            sum.x += (float)input[idx + ch * CHIRP_LEN * NUM_CHIRPS];
            sum.y += (float)input[idx + ch * CHIRP_LEN * NUM_CHIRPS];
        }
        output[idx].x = sum.x / NUM_CHANNELS;
        output[idx].y = sum.y / NUM_CHANNELS;
		
    }

	//printf(" %f, %f ", output[0].x, output[idx].y);
}

__global__ void applyLogScaling(float *input, float *output, int size) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = 10 * log10f(input[idx] + 1e-6);
    }
}

__global__ void computePower(float2 *d_avg, float *d_power) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < CHIRP_LEN * NUM_CHIRPS) {
        d_power[idx] = d_avg[idx].x * d_avg[idx].x + d_avg[idx].y * d_avg[idx].y;
    }
}

void configure_client_socket(){

    client_sockfd = socket(AF_INET, SOCK_STREAM, 0);
    
	if (client_sockfd < 0) {
        perror("\nSocket creation failed");
        return;
    }
	printf("\n Created the socket stream");
    
    out_server_addr.sin_family = AF_INET;
    out_server_addr.sin_port = htons(CLIENT_PORT);
    //out_server_addr.sin_addr.s_addr = inet_addr("192.168.33.29");

	printf("\nConfigured the address and port");

    if (inet_pton(AF_INET, "127.0.0.1", &out_server_addr.sin_addr) <= 0) {
        perror ("Invalid address/ Address not supported");
        return;
    }
    printf("Connecting to the socket ");
	if (connect(client_sockfd, (struct sockaddr*)&out_server_addr, sizeof(out_server_addr)) < 0) {
        perror("\nConnection failed 127.0.0.1 ");
        close(client_sockfd);
        return;
    }
	printf("\n Successfully connected to the client");
}




void close_client_socket(){
	close(client_sockfd);
}

void send_output(float *output, int size) {
	printf("\nSend output Size : %d ", size);

    send(client_sockfd, output, size * sizeof(float), 0);
    
	return;
}



void process_frame(short *h_input, int device_id) {

	printf("Process frame. Device Id : %d", device_id);

    //hipSetDevice(device_id);
	//hipDeviceSynchronize();

    short *d_input;
	float2 *d_avg;
    float *d_power, *d_log_power, *d_output;
    float *h_output = new float[CHIRP_LEN * NUM_CHIRPS];
   
	printf("\nAllocating memories. Device Id %d ", device_id);
    gpuErrchk(hipMalloc((void **)&d_input, FRAME_SIZE * sizeof(short) * 16));
    gpuErrchk(hipMalloc((void **)&d_avg, CHIRP_LEN * NUM_CHIRPS * sizeof(float2)*16));
    gpuErrchk(hipMalloc((void **)&d_power, CHIRP_LEN * NUM_CHIRPS * sizeof(float)*8));
    gpuErrchk(hipMalloc((void **)&d_log_power, CHIRP_LEN * NUM_CHIRPS * sizeof(float)*8));
    gpuErrchk(hipMalloc((void **)&d_output, CHIRP_LEN * NUM_CHIRPS * sizeof(float)));
  	int size = (int)FRAME_SIZE * sizeof(short)*16;
	printf("\nMemory allocation completed. Copying the input to h_input, Size  %d ", size);
    gpuErrchk(hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice));

	/*
	int i = 0, j = 0;
	int avg_index = 0;
	int base_index, chirp_index, channel_index;
	for ( i = 0; i < NUM_CHIRPS; i++ ){
		//base_index = i * CHIRP_LEN*2*4;				
		for (j = 0 ; j < CHIRP_LEN*2; j=j+2){
			base_index = i * CHIRP_LEN + j;
			d_avg[avg_index].x = d_input[base_index];
			d_avg[avg_index].y = d_input[base_index +1]; 
			printf("%f + j%f", d_avg[avg_index].x, d_avg[avg_index].y);
		}
	}
 	*/
	printf("\nCreating 3 dimensional array");
    
	dim3 blockSize(256);
	//printf("\nCreated 3 dim block");
	int grid_size = (CHIRP_LEN * NUM_CHIRPS + blockSize.x - 1) / blockSize.x;

	//printf("\nGrid size = %d", grid_size);
    dim3 gridSize(grid_size);
	printf("\nPopulating the average of all the channel data ");
    averageChannels<<<gridSize, blockSize>>>(d_input, d_avg);
	gpuErrchk(hipDeviceSynchronize());

	printf("\nAverages calculated %f %f %f %f", d_avg[0].x, d_avg[0].y, d_avg[1].x, d_avg[1].y);
	printf("\nAverages calculated" );

	if( d_input == NULL || d_avg == NULL)
		printf("\nCalulated the average failed");

	//printf("\nCalculated the average %f %f %f %f", d_avg[0].x,  d_avg[0].y,  d_avg[16].x, d_avg[16].x );  	
	
    hipfftHandle plan;
	printf("\nCalculating 2 dim fft. Device id = %d", device_id);
    hipfftPlan2d(&plan, CHIRP_LEN, NUM_CHIRPS, HIPFFT_C2C);
    hipfftExecC2C(plan, d_avg, d_avg, HIPFFT_FORWARD);
	
	computePower<<<gridSize, blockSize>>>(d_avg, d_power);

	printf("\nCompleted calculation of 2Dim FFT. Device id =%d", device_id);
	//if ( plan != NULL)
    hipfftDestroy(plan);

	//printf("\nComputing the Absolute value. Device Id = %d\n", device_id);

	//checkCudaErrors(hipGetLastError());
	//computePower<<<gridSize, blockSize>>>(d_avg, d_avg);
	
    printf("\nApplying the logerthemic scaling ");

    applyLogScaling<<<gridSize, blockSize>>>(d_power, d_log_power, CHIRP_LEN * NUM_CHIRPS);

    hipMemcpy(h_output, d_log_power, CHIRP_LEN * NUM_CHIRPS * sizeof(float), hipMemcpyDeviceToHost);

	printf("\nSending the output through socket. Device Id : %d. Length = %d", device_id, (CHIRP_LEN * NUM_CHIRPS));
    //send_output(h_output, CHIRP_LEN * NUM_CHIRPS);
	send_output(h_output, CHIRP_LEN * NUM_CHIRPS);
	printf("\nFreeing all the memories");
    hipFree(d_input);
    hipFree(d_avg);
    hipFree(d_power);
    hipFree(d_log_power);
    hipFree(d_output);
    delete[] h_output;

}

void configure_server_socket(){
    std::vector<std::thread> threads;
    int device_id = 0;
	int server_fd, new_socket;
	unsigned char buffer[BUFFER_SIZE];
	int i = 0;
	
	socklen_t addrlen = sizeof(in_server_addr);
	int opt = 1;
    if ((server_fd = socket(AF_INET, SOCK_STREAM, 0)) == 0) {
        perror("Socket failed");
        exit(EXIT_FAILURE);
    }
	printf("\nServer socket created");
    if (setsockopt(server_fd, SOL_SOCKET, SO_REUSEADDR, &opt, sizeof(opt))) {
        perror("setsockopt");
        close(server_fd);
        exit(EXIT_FAILURE);
    }
	printf("\nServer socket options are set");
    in_server_addr.sin_family = AF_INET;
    in_server_addr.sin_addr.s_addr = inet_addr(IN_SERVER_IP_ADDRESS);
    in_server_addr.sin_port = htons(IN_SERVER_PORT);
	printf("\nServer socket configured. Binding to the ports");

    if (bind(server_fd, (struct sockaddr *)&in_server_addr, sizeof(in_server_addr)) < 0) {
        perror("Bind failed");
        close(server_fd);
        exit(EXIT_FAILURE);
    }
	printf("\nBinding to the server socket successful. Listening");

    if (listen(server_fd, 3) < 0) {
        perror("Listen failed");
        close(server_fd);
        exit(EXIT_FAILURE);
    }

	printf("Server listening on %s:%d\n", IN_SERVER_IP_ADDRESS, IN_SERVER_PORT);
	
    for (i = 0; i < 1; i++ ) {
        // Accept a new connection
        if ((new_socket = accept(server_fd, (struct sockaddr *)&in_server_addr, &addrlen)) < 0) {
            perror("Accept failed");
            continue;
        }
        printf("Connected to client\n");
		int total_frame_bytes = FRAME_SIZE * sizeof(short)*16;
		//short *h_input = new short[FRAME_SIZE*16];
		short *h_input;
		short *d_input = new short[FRAME_SIZE*16];
		int current_pos = 0;
		int packet_size  = 1296 * 16;

    	while (true) {		   
			//int bytes_read = read(new_socket, reinterpret_cast<short*>(h_input), total_frame_bytes);
			
			h_input = new short[packet_size];
			int bytes_read = read(new_socket, h_input, packet_size);
			printf("\nRead %d %d %d %d ", h_input[0], h_input[1], h_input[2], h_input[3]);
			if (bytes_read > 0) {
		        //printf("Received %d bytes of binary data\n", bytes_read);
		        // Echo binary data back to client
		        //send(new_socket, buffer, bytes_read, 0);
				if( current_pos + bytes_read >= total_frame_bytes){
					int delta = total_frame_bytes - current_pos;
					
					printf("\nbytes read = %d, delta = %d", bytes_read, delta);
					printf("\nTotal allocated %d. Total captured %d", total_frame_bytes, (current_pos+bytes_read));
					hipMemcpy(&d_input[current_pos], h_input, bytes_read, hipMemcpyHostToDevice);
					break;
				}	
				
				hipMemcpy(&d_input[current_pos], h_input, bytes_read, hipMemcpyHostToDevice);
				current_pos = current_pos + bytes_read;
				//printf("Bytes read in this packet = %d. Total bytes read %d", bytes_read, current_pos);
				/*				
				if( current_pos >= total_frame_bytes){
					printf("\nReading of Frame bytes completed. Total read : %d \n", current_pos);
					break;
				} 
				*/
				delete h_input;
				h_input = new short[FRAME_SIZE*16];
		    }else{
				printf("Do not know why bytes are not read. Current size : %d", current_pos);
				break;			
			}

			//break;
		    //process_frame(h_input, device_id);
			//sleep(10);
			//device_id = (device_id + 1) % device_count;

    	}
		delete h_input;
		threads.emplace_back(process_frame, d_input, device_id);
		device_id = device_id + 1; 
        // Read binary data from client
        // Close client socket
        close(new_socket);
    }
    for (auto &t : threads) {
        t.join();
    }
	close_client_socket();
	close(server_fd);
}

int main() {

	int max_depth = 2;
	//cdp_kernel<<<4, 4>>>(max_depth, 0, 0, -1);
    checkCudaErrors(hipGetLastError());
    int device_count;
    hipGetDeviceCount(&device_count);
	printf("Maximumn number of devices = %d", device_count);
	

    //std::ifstream file("/home/nvdia/Programs/Services/bin/udp_received_file.bin", std::ios::binary);
	/*	
	std::ifstream file("/home/nvdia/UART_com/test.bin", std::ios::binary);

    if (!file) {
        printf("\n\nError: Unable to open file!\n");
        return -1;
    }*/
	printf("\n File opened ");

    //std::vector<std::thread> threads;
    //int device_id = 0;
	
 	configure_client_socket();
	printf("\nReading the file and pupulating the memory ");
	configure_server_socket();
	/*
    while (true) {
        float2 *h_input = new float2[FRAME_SIZE];
        if (!file.read(reinterpret_cast<char*>(h_input), FRAME_SIZE * sizeof(float2))) {
            delete[] h_input;
            break;
        }
		printf("\nAssigning each frame to a different thread");
        threads.emplace_back(process_frame, h_input, device_id);
		device_id = device_id + 1; 
		break;
        //process_frame(h_input, device_id);
		//sleep(10);
		//device_id = (device_id + 1) % device_count;
    }

    file.close();
	*/

	

	/*
    for (auto &t : threads) {
        t.join();
    }
	close_client_socket();
*/
    return 0;
}


